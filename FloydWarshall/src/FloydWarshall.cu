#include "hip/hip_runtime.h"

#include "FloydWarshall.cuh"
#include "Kernels.cuh"

// ----------------------------------------------------------------------------
// PERFORM PARALLEL FLOYD-WARSHALL
// ----------------------------------------------------------------------------
template <typename T>
float parallel_floyd_warshall(T* h_N, int n, int kernel_number, int threads_per_block) {
  float *d_N;
  int size = n * n * sizeof(float);

  hipEvent_t startTimeCuda, stopTimeCuda;
  hipEventCreate(&startTimeCuda);
  hipEventCreate(&stopTimeCuda);

  // 1. Allocate global memory on the device for N
  CHECK_ERROR(hipMalloc((void**)&d_N, size));

  // copy N to device memory
  CHECK_ERROR(hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice));

  dim3 dimGrid;
  dim3 dimBlock;

// For blocked algorithm
/******************************************************************************/
  int stages = ceil(n / (float)TILE_WIDTH);

  // dimensions
  dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 phase1Grid(1, 1, 1);
  dim3 phase2Grid(stages-1, 2, 1);
  dim3 phase3Grid(stages, stages, 1);
  //dim3 phase3Grid(stages-1, stages-1, 1);
/******************************************************************************/

  // printf("Grid:   {%d,\t%d,\t%d} blocks.\nBlocks: {%d,\t%d,\t%d} threads.\n", \
  //         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

  // 2. Kernel launch code - to have the device to perform the Floyd Warshall algorithm
  // ------------------- CUDA COMPUTATION ---------------------------
  hipEventRecord(startTimeCuda, 0);
  hipEventSynchronize(startTimeCuda);

  switch (kernel_number) {
    case 1:
      dimGrid  = dim3( ceil(n / (float)BLOCK_NAIVE), ceil(n / (float)BLOCK_NAIVE), 1.0);
      dimBlock = dim3(BLOCK_NAIVE, BLOCK_NAIVE,1.0);

      for (int k = 0; k < n; k++)
        naive_floyd_warshall_kernel <<< dimGrid, dimBlock >>> (d_N, n, k);
      break;
    case 2:
      //dimGrid = dim3( ceil(n / (float)BLOCK_SIZE), ceil(n / (float)BLOCK_SIZE), 1.0);
      dimGrid  = dim3(ceil((float)n*n/(BLOCK_COA*SEGMENT_SIZE)));
      dimBlock = dim3(BLOCK_COA, 1.0,1.0);

      for(int k = 0; k < n; ++k)
        coa_floyd_warshall_kernel<<<dimGrid, dimBlock>>>(d_N, n, k);
      break;
    case 3:
      dimGrid  = dim3(ceil(n / (float)BLOCK_SM), n, 1.0);
      dimBlock = dim3(BLOCK_SM, 1.0, 1.0);

      for(int k = 0; k < n; ++k)
        sm_floyd_warshall_kernel<<<dimGrid, dimBlock>>>(d_N, n, k);
      break;
    case 4:

      for(int k = 0; k < stages; k++) {
    		int base = TILE_WIDTH * k;
        phase1<<<phase1Grid, blockSize>>>(d_N, n, base);
        phase2<<<phase2Grid, blockSize>>>(d_N, n, k, base);
        phase3<<<phase3Grid, blockSize>>>(d_N, n, k, base);
      }

      break;
    default:
      break;
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel %d (error code %s)!\n", kernel_number, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipEventRecord(stopTimeCuda, 0);
  hipEventSynchronize(stopTimeCuda);


  // ---------------------- CUDA ENDING -----------------------------
  float msTime;
  hipEventElapsedTime(&msTime, startTimeCuda, stopTimeCuda);
  printf("DeviceTime: %f\n\n", msTime);

  // 3. copy result from the device memory
  CHECK_ERROR(hipMemcpy(h_N, d_N, size, hipMemcpyDeviceToHost));

  // cleanup memory
  CHECK_ERROR(hipFree(d_N));

  return msTime;
}

template float parallel_floyd_warshall<float>(float*, int, int, int);
