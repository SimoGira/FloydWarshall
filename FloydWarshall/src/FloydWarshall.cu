#include "hip/hip_runtime.h"

#pragma once
#include "FloydWarshall.cuh"
#include "Kernels.cuh"

// ----------------------------------------------------------------------------
// PERFORM PARALLEL FLOYD-WARSHALL
// ----------------------------------------------------------------------------
template <typename T>
void parallel_floyd_warshall(T* h_N, int n, int kernel_number) {
  printf("Called parallel_floyd_warshall\n");

  float *d_N;
  int size = n * n * sizeof(float);

  // hipEvent_t startTimeCuda, stopTimeCuda;
  // hipEventCreate(&startTimeCuda);
  // hipEventCreate(&stopTimeCuda);

  // 1. Allocate global memory on the device for N
  CHECK_ERROR(hipMalloc((void**)&d_N, size));

  // copy N to device memory
  CHECK_ERROR(hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice));

  // copy infinty constant to constant memory
  //CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dest), source, size));

  dim3 dimGrid(ceil(n / (float)BLOCK_SIZE), ceil(n / (float)BLOCK_SIZE), 1);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1.0);

  printf("Grid:   {%d,\t%d,\t%d} blocks.\nBlocks: {%d,\t%d,\t%d} threads.\n", \
          dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

  // 2. Kernel launch code - to have the device to perform the Floyd Warshall algorithm
  // ------------------- CUDA COMPUTATION ---------------------------
  // hipEventRecord(startTimeCuda, 0);
  // hipEventSynchronize(startTimeCuda);

  switch (kernel_number) {
    case 1:
      for (int k = 0; k < n; k++) {
        naive_floyd_warshall_kernel <<< dimGrid, dimBlock >>> (d_N, n, k);
      }
      break;
    case 2:
      dim3 dimGrid(n, n, 1);                      // < ---- check!!!
      for(int k = 0; k < n; ++k)
        coa_floyd_warshall_kernel<<<dimGrid, 1>>>(d_N, n, k);
      break;
    case 3:
      dim3 dimGrid(n, n / TILE_WIDTH, 1);
      dim3 dimBlock(1, TILE_WIDTH, 1);
      for(int k = 0; k < n; ++k)
        sm_floyd_warshall_kernel<<<dimGrid, dimBlock>>>(d_N, n, k);
      break;
    case 4:
      // TODO Blocked_kernel
      break;
    default:
      break;
  }

  // hipEventRecord(stopTimeCuda, 0);
  // hipEventSynchronize(stopTimeCuda);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel %d (error code %s)!\n", kernel_number, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ---------------------- CUDA ENDING -----------------------------
  // float msTime;
  // hipEventElapsedTime(&msTime, startTimeCuda, stopTimeCuda);
  // printf("KernelTime: %fn", msTime);

  // 3. copy result from the device memory
  CHECK_ERROR(hipMemcpy(h_N, d_N, size, hipMemcpyDeviceToHost));

  // // cleanup memory
  CHECK_ERROR(hipFree(d_N));

  printf("return from parallel_floyd_warshall\n\n");
  // return msTime;
}

template void parallel_floyd_warshall<float>(float*, int, int);
